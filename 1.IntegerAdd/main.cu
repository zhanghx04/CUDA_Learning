#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void AddIntsCUDA(int* a, int* b)
{
    a[0] += b[0];
}

int main()
{
    int a = 5;
    int b = 9;
    
    int *d_a, *d_b;
    
    // Malloc space for device variables
    hipMalloc(&d_a, sizeof(int));
    hipMalloc(&d_b, sizeof(int));
    
    // Copy memory to device
    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);
    
    AddIntsCUDA<<<1, 1>>>(d_a, d_b);
    
    // copy the value from device to host back
    hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost);
    
    cout << "The answer is " << a << endl;
    
    // Release the memory
    hipFree(d_a);
    hipFree(d_b);
    
    return 0;
}
