
#include <hip/hip_runtime.h>
#include <iostream>  
#include <fstream>   

using namespace std;

#define BLOCK_WIDTH 16 
#define BLOCK_HEIGHT 16

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) 
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

/**********************************************/
/* KERNEL WITH OPTIMIZED USE OF SHARED MEMORY */
/**********************************************/
__global__ void Optimized_Kernel_Function_shared(unsigned short *Input_Image, unsigned short *Output_Image, int Image_Width, int Image_Height)
{
    const int tx_l = threadIdx.x;                           // --- Local thread x index
    const int ty_l = threadIdx.y;                           // --- Local thread y index

    const int tx_g = blockIdx.x * blockDim.x + tx_l;        // --- Global thread x index
    const int ty_g = blockIdx.y * blockDim.y + ty_l;        // --- Global thread y index

    __shared__ unsigned short smem[BLOCK_WIDTH+2][BLOCK_HEIGHT+2];

    // --- Fill the shared memory border with zeros
    if (tx_l == 0)                      smem[tx_l]  [ty_l+1]    = 0;    // --- left border
    else if (tx_l == BLOCK_WIDTH-1)     smem[tx_l+2][ty_l+1]    = 0;    // --- right border
    if (ty_l == 0) {                    smem[tx_l+1][ty_l]      = 0;    // --- upper border
        if (tx_l == 0)                  smem[tx_l]  [ty_l]      = 0;    // --- top-left corner
        else if (tx_l == BLOCK_WIDTH-1) smem[tx_l+2][ty_l]      = 0;    // --- top-right corner
    } else if (ty_l == BLOCK_HEIGHT-1) {smem[tx_l+1][ty_l+2]  = 0;    // --- bottom border
      if (tx_l == 0)                  smem[tx_l]  [ty_l+2]    = 0;    // --- bottom-left corder
        else if (tx_l == BLOCK_WIDTH-1) smem[tx_l+2][ty_l+2]    = 0;    // --- bottom-right corner
    }

    // --- Fill shared memory
                                                                        smem[tx_l+1][ty_l+1] = Input_Image[ty_g*Image_Width + tx_g];      // --- center
    if ((tx_l == 0)&&((tx_g > 0)))                                      smem[tx_l]  [ty_l+1] = Input_Image[ty_g*Image_Width + tx_g-1];      // --- left border
    else if ((tx_l == BLOCK_WIDTH-1)&&(tx_g < Image_Width - 1))         smem[tx_l+2][ty_l+1] = Input_Image[ty_g*Image_Width + tx_g+1];      // --- right border
    if ((ty_l == 0)&&(ty_g > 0)) {                                      smem[tx_l+1][ty_l]   = Input_Image[(ty_g-1)*Image_Width + tx_g];    // --- upper border
        if ((tx_l == 0)&&((tx_g > 0)))                                  smem[tx_l]  [ty_l]   = Input_Image[(ty_g-1)*Image_Width + tx_g-1];  // --- top-left corner
        else if ((tx_l == BLOCK_WIDTH-1)&&(tx_g < Image_Width - 1))     smem[tx_l+2][ty_l]   = Input_Image[(ty_g-1)*Image_Width + tx_g+1];  // --- top-right corner
    } else if ((ty_l == BLOCK_HEIGHT-1)&&(ty_g < Image_Height - 1)) {  smem[tx_l+1][ty_l+2] = Input_Image[(ty_g+1)*Image_Width + tx_g];    // --- bottom border
        if ((tx_l == 0)&&((tx_g > 0)))                                 smem[tx_l]  [ty_l+2] = Input_Image[(ty_g-1)*Image_Width + tx_g-1];  // --- bottom-left corder
        else if ((tx_l == BLOCK_WIDTH-1)&&(tx_g < Image_Width - 1))     smem[tx_l+2][ty_l+2] = Input_Image[(ty_g+1)*Image_Width + tx_g+1];  // --- bottom-right corner
    }
    __syncthreads();

    // --- Pull the 3x3 window in a local array
    unsigned short v[9] = { smem[tx_l][ty_l],   smem[tx_l+1][ty_l],     smem[tx_l+2][ty_l],
                            smem[tx_l][ty_l+1], smem[tx_l+1][ty_l+1],   smem[tx_l+2][ty_l+1],
                            smem[tx_l][ty_l+2], smem[tx_l+1][ty_l+2],   smem[tx_l+2][ty_l+2] };    

    // --- Bubble-sort
    for (int i = 0; i < 5; i++) {
        for (int j = i + 1; j < 9; j++) {
            if (v[i] > v[j]) { // swap?
                unsigned short tmp = v[i];
                v[i] = v[j];
                v[j] = tmp;
            }
         }
    }

    // --- Pick the middle one
    Output_Image[ty_g*Image_Width + tx_g] = v[4];
}

/****************************/
/* ORIGINAL KERNEL FUNCTION */
/****************************/
__global__ void Original_Kernel_Function(unsigned short *Input_Image, unsigned short *Output_Image, int Image_Width, int Image_Height) {

    __shared__ unsigned short surround[BLOCK_WIDTH*BLOCK_HEIGHT][9];

    int iterator;

    const int x     = blockDim.x * blockIdx.x + threadIdx.x;
    const int y     = blockDim.y * blockIdx.y + threadIdx.y;
    const int tid   = threadIdx.y * blockDim.x + threadIdx.x;   

    if( (x >= (Image_Width - 1)) || (y >= Image_Height - 1) || (x == 0) || (y == 0)) return;

    // --- Fill shared memory
    iterator = 0;
    for (int r = x - 1; r <= x + 1; r++) {
        for (int c = y - 1; c <= y + 1; c++) {
            surround[tid][iterator] = Input_Image[c*Image_Width+r];
            iterator++;
        }
    }

    // --- Sort shared memory to find the median using Bubble Short
    for (int i=0; i<5; ++i) {

        // --- Find the position of the minimum element
        int minval=i;
        for (int l=i+1; l<9; ++l) if (surround[tid][l] < surround[tid][minval]) minval=l;

        // --- Put found minimum element in its place
        unsigned short temp = surround[tid][i];
        surround[tid][i]=surround[tid][minval];
        surround[tid][minval]=temp;
    }

    // --- Pick the middle one
    Output_Image[(y*Image_Width)+x]=surround[tid][4]; 

    __syncthreads();

}

/***********************************************/
/* ORIGINAL KERNEL FUNCTION - NO SHARED MEMORY */
/***********************************************/
__global__ void Original_Kernel_Function_no_shared(unsigned short *Input_Image, unsigned short *Output_Image, int Image_Width, int Image_Height) {

    unsigned short surround[9];

    int iterator;

    const int x     = blockDim.x * blockIdx.x + threadIdx.x;
    const int y     = blockDim.y * blockIdx.y + threadIdx.y;
    const int tid   = threadIdx.y * blockDim.x + threadIdx.x;   

    if( (x >= (Image_Width - 1)) || (y >= Image_Height - 1) || (x == 0) || (y == 0)) return;

    // --- Fill array private to the threads
    iterator = 0;
    for (int r = x - 1; r <= x + 1; r++) {
        for (int c = y - 1; c <= y + 1; c++) {
            surround[iterator] = Input_Image[c*Image_Width+r];
            iterator++;
        }
    }

    // --- Sort private array to find the median using Bubble Short
    for (int i=0; i<5; ++i) {

        // --- Find the position of the minimum element
        int minval=i;
        for (int l=i+1; l<9; ++l) if (surround[l] < surround[minval]) minval=l;

        // --- Put found minimum element in its place
        unsigned short temp = surround[i];
        surround[i]=surround[minval];
        surround[minval]=temp;
    }

    // --- Pick the middle one
    Output_Image[(y*Image_Width)+x]=surround[4]; 

}

/********/
/* MAIN */
/********/
int main()
{
    const int Image_Width = 1580;
    const int Image_Height = 1050;

    // --- Open data file
    ifstream is; is.open("C:\\Users\\user\\Documents\\Project\\Median_Filter\\Release\\Image_To_Be_Filtered.raw", ios::binary );

    // --- Get file length
    is.seekg(0, ios::end);
    int dataLength = is.tellg();
    is.seekg(0, ios::beg);

    // --- Read data from file and close file
    unsigned short* Input_Image_Host = new unsigned short[dataLength * sizeof(char) / sizeof(unsigned short)];
    is.read((char*)Input_Image_Host,dataLength);
    is.close();

    // --- CUDA warm up
    unsigned short *forFirstCudaMalloc; gpuErrchk(hipMalloc((void**)&forFirstCudaMalloc, dataLength * sizeof(unsigned short)));
    gpuErrchk(hipFree(forFirstCudaMalloc));

    // --- Allocate host and device memory spaces 
    unsigned short *Output_Image_Host = (unsigned short *)malloc(dataLength);
    unsigned short *Input_Image; gpuErrchk(hipMalloc( (void**)&Input_Image, dataLength * sizeof(unsigned short))); 
    unsigned short *Output_Image; gpuErrchk(hipMalloc((void**)&Output_Image, dataLength * sizeof(unsigned short))); 

    // --- Copy data from host to device
    gpuErrchk(hipMemcpy(Input_Image, Input_Image_Host, dataLength, hipMemcpyHostToDevice));// copying Host Data To Device Memory For Filtering

    // --- Grid and block sizes
    const dim3 grid (iDivUp(Image_Width, BLOCK_WIDTH), iDivUp(Image_Height, BLOCK_HEIGHT), 1);      
    const dim3 block(BLOCK_WIDTH, BLOCK_HEIGHT, 1); 

    /****************************/
    /* ORIGINAL KERNEL FUNCTION */
    /****************************/
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(Original_Kernel_Function), hipFuncCachePreferShared);
    Original_Kernel_Function<<<grid,block>>>(Input_Image, Output_Image, Image_Width, Image_Height);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Original kernel function - elapsed time:  %3.3f ms \n", time);

    /***********************************************/
    /* ORIGINAL KERNEL FUNCTION - NO SHARED MEMORY */
    /***********************************************/
    hipEventRecord(start, 0);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(Original_Kernel_Function_no_shared), hipFuncCachePreferL1);
    Original_Kernel_Function_no_shared<<<grid,block>>>(Input_Image, Output_Image, Image_Width, Image_Height);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Original kernel function - no shared - elapsed time:  %3.3f ms \n", time);

    /**********************************************/
    /* KERNEL WITH OPTIMIZED USE OF SHARED MEMORY */
    /**********************************************/
    hipEventRecord(start, 0);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(Optimized_Kernel_Function_shared), hipFuncCachePreferShared);
    Optimized_Kernel_Function_shared<<<grid,block>>>(Input_Image, Output_Image, Image_Width, Image_Height);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Optimized kernel function - shared - elapsed time:  %3.3f ms \n", time);

    // --- Copy results back to the host
    gpuErrchk(hipMemcpy(Output_Image_Host, Output_Image, dataLength, hipMemcpyDeviceToHost));

    // --- Open results file, write results and close the file
    ofstream of2;     of2.open("C:\\Users\\angelo\\Documents\\Project\\Median_Filter\\Release\\Filtered_Image.raw",  ios::binary);
    of2.write((char*)Output_Image_Host, dataLength);
    of2.close();

    cout << "\n Press Any Key To Exit..!!";
    gpuErrchk(hipFree(Input_Image));

    delete Input_Image_Host;
    delete Output_Image_Host;

    return 0;
}
